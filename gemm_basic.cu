#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>

#include "util.cuh"

namespace {
__global__ void gemmKernel(const float *__restrict__ A,
                             const float *__restrict__ B,
                             float *__restrict__ C,
                             unsigned M, unsigned N, unsigned K) {
    unsigned c = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned r = blockIdx.y * blockDim.y + threadIdx.y;
    if (c >= N || r >= M) {
        return ;
    }
    Tensor2D<const float> tensorA = Tensor2D<const float>(A, M, K);
    Tensor2D<const float> tensorB = Tensor2D<const float>(B, K, N);
    Tensor2D<float> tensorC = Tensor2D<float>(C, M, N);
    float sum = 0.0;
    for (unsigned i = 0; i < K; i++) {
        sum += tensorA[r][i] * tensorB[i][c];
    }
    tensorC[r][c] = sum;
}
}

void gemmBasic(const float *__restrict__ A, const float *__restrict__ B, float *__restrict__ C,
            unsigned M, unsigned N, unsigned K) {
    // Device malloc
    float *d_x, *d_y, *d_z;
    hipMalloc(&d_x, M*K*sizeof(float));
    hipMalloc(&d_y, K*N*sizeof(float));
    hipMalloc(&d_z, M*N*sizeof(float));

    // Host to device
    hipMemcpy(d_x, A, M*K*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, B, K*N*sizeof(float), hipMemcpyHostToDevice);

    // invoke
    dim3 dimBlock(16, 16);
    dim3 dimGrid((N-1) / dimBlock.x + 1, (M-1) / dimBlock.y + 1);
    gemmKernel<<<dimGrid, dimBlock>>>(d_x, d_y, d_z, M, N, K);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Device to host
    hipMemcpy(C, d_z, M*N*sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
}